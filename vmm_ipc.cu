#include <iostream>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include "cuutils.h"

#define CUUTIL_DEBUG
#define SIZE_TO_ALLOC 1024
#define ROUND_UP(x, m) ((m) > 0 ? (((x) + (m) - 1) / (m)) * (m) : (x))

void * ec_malloc(size_t sz) {
    void * ptr = nullptr;
    if((ptr = (void *)malloc(sz)) == nullptr) {
        std::cout << "malloc failed" << std::endl;
    }
    return ptr;
}


int main() {
    int device_id = 0;
    int supportsVMM = 0;
    CUUTIL_ERRCHK(hipFree(0));  // Force and check the initialization of the runtime

    CUUTIL_ERRCHK(hipCtxGetDevice(&device_id));
    std::cout << "device id = " << device_id << std::endl;
    CUUTIL_ERRCHK(hipDeviceGetAttribute(&supportsVMM, CU_DEVICE_ATTRIBUTE_VIRTUAL_ADDRESS_MANAGEMENT_SUPPORTED, device_id));
    
    size_t granularity = 0;
    hipMemGenericAllocationHandle_t allocHandle;
    hipMemAccessDesc accessDesc;
    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device_id;
    accessDesc.location = prop.location;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    hipDeviceptr_t ptr;

    CUUTIL_ERRCHK(hipMemGetAllocationGranularity(&granularity, &prop,
                                            hipMemAllocationGranularityMinimum));
    size_t padded_size = ROUND_UP(SIZE_TO_ALLOC, granularity);
    CUUTIL_ERRCHK(hipMemCreate(&allocHandle, padded_size, &prop, 0));

    std::cout << "granularity = " << granularity << ", padded_size = " << padded_size << std::endl;
    CUUTIL_ERRCHK(hipMemAddressReserve(&ptr, padded_size, 0, 0, 0));    
    std::cout << "Reserved VA = 0x" << std::hex << ptr << std::endl;

    CUUTIL_ERRCHK(hipMemMap(ptr, padded_size, 0, allocHandle, 0));
    CUUTIL_ERRCHK(hipMemSetAccess(ptr, padded_size, &accessDesc, 1));

    CUUTIL_ERRCHK(hipMemset((void *)ptr, (int)'A', padded_size));

    char * host_src = (char *)ec_malloc(padded_size);
    char * host_dst = (char *)ec_malloc(padded_size);

    memset(host_src, 'S', padded_size);
    memset(host_dst, 'D', padded_size);
    CUUTIL_ERRCHK(hipMemcpy((void *)ptr, (void *)host_src, padded_size, hipMemcpyHostToDevice));
    CUUTIL_ERRCHK(hipMemcpy((void *)host_dst, (void *)ptr, padded_size, hipMemcpyDeviceToHost));
    std::cout << "peeking host dst" << std::endl;
    std::cout << host_dst[0] << host_dst[1] << host_dst[2] << host_dst[3] << std::endl;
    
    CUUTIL_ERRCHK(hipMemUnmap(ptr, padded_size));
    CUUTIL_ERRCHK(hipMemRelease(allocHandle));
    CUUTIL_ERRCHK(hipMemAddressFree(ptr, padded_size));
    free(host_src);
    free(host_dst);
    return 0;
}