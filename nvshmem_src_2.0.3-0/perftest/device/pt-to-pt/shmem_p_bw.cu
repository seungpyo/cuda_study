#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <getopt.h>
#include "utils.h"

#define MAX_ITERS 10
#define MAX_SKIP 10
#define THREADS 1024
#define BLOCKS 4
#define MAX_MSG_SIZE 64 * 1024
#define UNROLL 2

__global__ void bw(volatile double *data_d, volatile unsigned int *counter_d, int len, int pe,
                   int iter, int skip, double *bw_result) {
    int u, i, j, peer, tid, slice;
    unsigned int counter;
    long long int start = 0, stop = 0;
    double time = 0;
    int threads = gridDim.x * blockDim.x;
    tid = blockIdx.x * blockDim.x + threadIdx.x;

    peer = !pe;
    slice = UNROLL * threads;

    for (i = 0; i < (iter + skip); i++) {
        if (i == skip) {
            nvshmem_quiet();
            start = clock64();
        }

        for (j = 0; j < len - slice; j += slice) {
            for (u = 0; u < UNROLL; ++u) {
                int idx = j + u * threads + tid;
                nvshmem_double_p((double *)data_d + idx, *(data_d + idx), peer);
            }
            __syncthreads();
        }

        for (u = 0; u < UNROLL; ++u) {
            int idx = j + u * threads + tid;
            if (idx < len) nvshmem_double_p((double *)data_d + idx, *(data_d + idx), peer);
        }

        // synchronizing across blocks
        __syncthreads();

        if (!threadIdx.x) {
            __threadfence();
            counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
            if (counter == (gridDim.x * (i + 1) - 1)) {
                *(counter_d + 1) += 1;
            }
            while (*(counter_d + 1) != i + 1)
                ;
        }

        __syncthreads();
    }

    // synchronizing across blocks
    __syncthreads();

    if (!threadIdx.x) {
        __threadfence();
        counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
        if (counter == (gridDim.x * (i + 1) - 1)) {
            nvshmem_quiet();
            *(counter_d + 1) += 1;
        }
        while (*(counter_d + 1) != i + 1)
            ;
    }

    __syncthreads();

    stop = clock64();
    time = (stop - start);

    if (!threadIdx.x && !blockIdx.x) {
        *bw_result = ((float)iter * (float)len * sizeof(double) * clockrate) / ((time / 1000) * 1024 * 1024 * 1024);
    }
}

int main(int argc, char *argv[]) {
    int mype, npes;
    double *data_d = NULL;
    unsigned int *counter_d;
    int max_blocks = BLOCKS, max_threads = THREADS;
    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_bw;

    int iter = MAX_ITERS;
    int skip = MAX_SKIP;
    int max_msg_size = MAX_MSG_SIZE;

    init_wrapper(&argc, &argv);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    while (1) {
        int c;
        c = getopt(argc, argv, "c:t:h");
        if (c == -1) break;

        switch (c) {
            case 'c':
                max_blocks = strtol(optarg, NULL, 0);
                break;
            case 't':
                max_threads = strtol(optarg, NULL, 0);
                break;
            default:
            case 'h':
                printf("-c [CTAs] -t [THREADS] \n");
                goto finalize;
        }
    }

    array_size = floor(log2((float)max_msg_size)) + 1;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_bw = (double *)h_tables[1];

    data_d = (double *)nvshmem_malloc(max_msg_size);
    CUDA_CHECK(hipMemset(data_d, 0, max_msg_size));

    CUDA_CHECK(hipMalloc((void **)&counter_d, sizeof(unsigned int) * 2));
    CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

    CUDA_CHECK(hipDeviceSynchronize());

    int size;
    i = 0;
    if (mype == 0) {
        for (size = 1024; size <= MAX_MSG_SIZE; size *= 2) {
            int blocks = max_blocks, threads = max_threads;
            h_size_arr[i] = size;
            CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

            bw<<<blocks, threads>>>(data_d, counter_d, size / sizeof(double), mype, iter, skip, &h_bw[i]);
            CUDA_CHECK(hipGetLastError());

            CUDA_CHECK(hipDeviceSynchronize());

            nvshmem_barrier_all();
            i++;
        }
    } else {
        for (size = 1024; size <= MAX_MSG_SIZE; size *= 2) {
            nvshmem_barrier_all();
        }
    }

    if (mype == 0) {
        print_table("shmem_p_bw", "None", "size (Bytes)", "BW", "GB/sec", '+', h_size_arr, h_bw, i);
    }

finalize:

    if (data_d) nvshmem_free(data_d);
    free_tables(h_tables, 2);
    finalize_wrapper();

    return 0;
}
