#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include "utils.h"

#define THREADS 512
#define MAX_MSG_SIZE 64 * 1024
#define UNROLL 8

__global__ void pull(volatile int *data_d, int len, int pe, int iter, int skip, double *lat_result) {
    long long int start, stop;
    double time;
    int i, j, tid, peer;

    peer = !pe;
    tid = threadIdx.x;

    for (i = 0; i < (iter + skip); i++) {
        if (i == skip) start = clock64();

        if (!pe) {
            for (j = 0; j < len; j += THREADS) {
                if (j + tid < len)
                    *(data_d + j + tid) = nvshmem_int_g((int *)data_d + j + tid, peer);
            }

            __syncthreads();

        }
    }
    stop = clock64();

    if ((pe == 0) && !tid) {
        time = (stop - start) / iter;
        *lat_result = time * 1000 / clockrate;
    }
}

int main(int c, char *v[]) {
    int mype, npes, size;
    int *data_d = NULL;

    int iter = 200;
    int skip = 20;
    int max_msg_size = MAX_MSG_SIZE;

    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_lat;

    init_wrapper(&c, &v);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    array_size = floor(log2((float)max_msg_size)) + 1;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_lat = (double *)h_tables[1];

    data_d = (int *)nvshmem_malloc(max_msg_size);
    CUDA_CHECK(hipMemset(data_d, 0, max_msg_size));

    nvshmem_barrier_all();

    CUDA_CHECK(hipDeviceSynchronize());

    i = 0;
    for (size = sizeof(int); size <= max_msg_size; size *= 2) {
        int nelems;
        h_size_arr[i] = size;
        nelems = size / sizeof(int);

        pull<<<1, THREADS>>>(data_d, nelems, mype, iter, skip, &h_lat[i]);

        CUDA_CHECK(hipDeviceSynchronize());
        nvshmem_barrier_all();
        i++;
    }

    if (mype == 0) {
        print_table("shmem_g_latency", "None", "size (Bytes)", "latency", "us", '-', h_size_arr, h_lat, i);
    }

finalize:

    if (data_d) nvshmem_free(data_d);
    free_tables(h_tables, 2);
    finalize_wrapper();

    return 0;
}
