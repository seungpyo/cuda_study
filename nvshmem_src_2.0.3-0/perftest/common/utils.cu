/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include "utils.h"

double *d_latency = NULL;
double *d_avg_time = NULL;
double *latency = NULL;
double *avg_time = NULL;
int mype = 0;
int npes = 0;
int use_mpi = 0;
int use_shmem = 0;
__device__ int clockrate;

void select_device() {
    hipDeviceProp_t prop;
    int dev_count;
    int mype_node;

    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);

    CUDA_CHECK(hipGetDeviceCount(&dev_count));
    CUDA_CHECK(hipSetDevice(mype_node % dev_count));

    CUDA_CHECK(hipGetDeviceProperties(&prop, mype_node % dev_count));
    fprintf(stderr, "mype: %d device name: %s bus id: %d \n", mype_node, prop.name, prop.pciBusID);
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(clockrate), (void *)&prop.clockRate, sizeof(int), 0,
                                  hipMemcpyHostToDevice));
}

void init_wrapper(int *c, char ***v) {
    char *value;

#ifdef NVSHMEM_MPI_SUPPORT
    value = getenv("NVSHMEMTEST_USE_MPI_LAUNCHER");
    if (value) use_mpi = atoi(value);
#endif

#ifdef NVSHMEM_SHMEM_SUPPORT
    value = getenv("NVSHMEMTEST_USE_SHMEM_LAUNCHER");
    if (value) use_shmem = atoi(value);
#endif

#ifdef NVSHMEM_MPI_SUPPORT
    if (use_mpi) {
        MPI_Init(c, v);
        int rank, nranks;
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        MPI_Comm_size(MPI_COMM_WORLD, &nranks);
        DEBUG_PRINT("MPI: [%d of %d] hello MPI world! \n", rank, nranks);
        MPI_Comm mpi_comm = MPI_COMM_WORLD;

        nvshmemx_init_attr_t attr;
        attr.mpi_comm = &mpi_comm;
        nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

        select_device();

        return;
    }
#endif

#ifdef NVSHMEM_SHMEM_SUPPORT
    if (use_shmem) {
        shmem_init();
        mype = shmem_my_pe();
        npes = shmem_n_pes();
        DEBUG_PRINT("SHMEM: [%d of %d] hello SHMEM world! \n", my_pe, n_pes);

        latency = (double *)shmem_malloc(sizeof(double));
        if (!latency) ERROR_EXIT("(shmem_malloc) failed \n");

        avg_time = (double *)shmem_malloc(sizeof(double));
        if (!avg_time) ERROR_EXIT("(shmem_malloc) failed \n");

        nvshmemx_init_attr_t attr;
        nvshmemx_init_attr(NVSHMEMX_INIT_WITH_SHMEM, &attr);

        select_device();

        return;
    }
#endif

    nvshmem_init();

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();
    select_device();

    d_latency = (double *)nvshmem_malloc(sizeof(double));
    if (!d_latency) ERROR_EXIT("nvshmem_malloc failed \n");

    d_avg_time = (double *)nvshmem_malloc(sizeof(double));
    if (!d_avg_time) ERROR_EXIT("nvshmem_malloc failed \n");

    DEBUG_PRINT("end of init \n");
    return;
}

void finalize_wrapper() {
#ifdef NVSHMEM_SHMEM_SUPPORT
    if (use_shmem) {
        shmem_free(latency);
        shmem_free(avg_time);
    }
#endif

#if !defined(NVSHMEM_SHMEM_SUPPORT) && !defined(NVSHMEM_MPI_SUPPORT)
    if (!use_mpi && !use_shmem) {
        nvshmem_free(d_latency);
        nvshmem_free(d_avg_time);
    }
#endif
    nvshmem_finalize();

#ifdef NVSHMEM_MPI_SUPPORT
    if (use_mpi) MPI_Finalize();
#endif
#ifdef NVSHMEM_SHMEM_SUPPORT
    if (use_shmem) shmem_finalize();
#endif
}

void
alloc_tables(void ***table_mem, int num_tables, int num_entries_per_table)
{
    void **tables;
    int i, dev_property;

    CUDA_CHECK(hipDeviceGetAttribute(&dev_property, hipDeviceAttributeUnifiedAddressing, nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE)));
    assert(dev_property == 1);

    assert(num_tables >= 1);
    assert(num_entries_per_table >= 1);
    CUDA_CHECK(hipHostAlloc(table_mem, num_tables * sizeof(void *), hipHostMallocMapped));
    tables = *table_mem;

    /* Just allocate an array of 8 byte values. The user can decide if they want to use double or uint64_t */
    for (i = 0; i < num_tables; i++) {
        CUDA_CHECK(hipHostAlloc(&tables[i], num_entries_per_table * sizeof(double), hipHostMallocMapped));
    }
}

void
free_tables(void **tables, int num_tables)
{
    int i;
    for (i = 0; i < num_tables; i++) {
        CUDA_CHECK(hipHostFree(tables[i]));
    }
    CUDA_CHECK(hipHostFree(tables));
}

void
print_table(const char *job_name, const char *subjob_name, const char *var_name,
            const char *output_var, const char *units, const char plus_minus,
            uint64_t *size, double *value, int num_entries)
{
	int i;

/* Used for automated test output. It outputs the data in a non human-friendly format. */
#ifdef NVSHMEM_MACHINE_READABLE_OUTPUT

    printf("%s\n", job_name);
    for (i = 0; i < num_entries; i++) {
        if (size[i] != 0 && value[i] != 0.00) {
            printf("&&&& PERF %s___%s___size__%lu___%s %lf %c%s\n", job_name, subjob_name, size[i], output_var, value[i], plus_minus, units);
        }
    }
#else
	printf("+------------------------+----------------------+\n");
	printf("| %-22s | %-20s |\n", job_name, subjob_name);
	printf("+------------------------+----------------------+\n");
	printf("| %-22s | %10s %-9s |\n", var_name, output_var, units);
	printf("+------------------------+----------------------+\n");
	for (i = 0; i < num_entries; i++) {
		printf("| %-22.0lu | %-20.6lf |\n", size[i], value[i]);
		printf("+------------------------+----------------------+\n");
	}
#endif
	printf("\n\n");
}
