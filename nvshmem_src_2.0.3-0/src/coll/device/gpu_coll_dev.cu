#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * See COPYRIGHT for license information
 */

#include "nvshmem.h"
#include "nvshmemx_error.h"
#include "util.h"
#include "gpu_coll.h"
#include "nvshmem_internal.h"

__device__ gpu_coll_env_params_t gpu_coll_env_params_var_d;


__device__ int reduce_recexch_step1_sendto_d;
__device__ int *reduce_recexch_step1_recvfrom_d;
__device__ int reduce_recexch_step1_nrecvs_d;
__device__ int **reduce_recexch_step2_nbrs_d;
__device__ int reduce_recexch_step2_nphases_d;
__device__ int reduce_recexch_p_of_k_d;
__device__ int reduce_recexch_reduce_recexch_digit_d;
__device__ int *digit_d;

extern "C" int init_shm_kernel_shm_ptr() {
    int status = 0;

    int *step1_recvfrom = NULL, **step2_nbrs = NULL;
    int *digit = NULL;
    int k, max_phases;

    status = hipMemcpyToSymbol(HIP_SYMBOL(gpu_coll_env_params_var_d), &gpu_coll_env_params_var,
                                sizeof(gpu_coll_env_params_t));
    NE_ERROR_JMP(status, hipSuccess, NVSHMEMX_ERROR_INTERNAL, out, "memcopy to symbol failed \n");

    /* Allocate memory for performing reduce recursive exchange algorithm */
    k = gpu_coll_env_params_var.reduce_recexch_kval;
    assert(k > 1);

    status = hipMalloc((hipDeviceptr_t *) &step1_recvfrom, sizeof(int) * (k - 1));
    NE_ERROR_JMP(status, hipSuccess, NVSHMEMX_ERROR_INTERNAL, out, "hipMalloc failed\n");

    status = hipMemcpyToSymbol(HIP_SYMBOL(reduce_recexch_step1_recvfrom_d), &step1_recvfrom, sizeof(void *));
    NE_ERROR_JMP(status, hipSuccess, NVSHMEMX_ERROR_INTERNAL, out, "memcopy to symbol failed \n");

    max_phases = log(nvshmemi_state->npes) / log(k) + 1; /* The '+ 1' makes it a conservative calculation, max_pahses >= 1 */

    status = hipMalloc((hipDeviceptr_t *) &step2_nbrs, sizeof(int *) * max_phases);
    NE_ERROR_JMP(status, hipSuccess, NVSHMEMX_ERROR_INTERNAL, out, "hipMalloc failed\n");
    status = hipMalloc((hipDeviceptr_t *) &digit, sizeof(int) * max_phases);
    NE_ERROR_JMP(status, hipSuccess, NVSHMEMX_ERROR_INTERNAL, out, "hipMalloc failed\n");

    for (int i = 0; i < max_phases; i++) {
        void *dev_ptr;
        status = hipMalloc(&dev_ptr, sizeof(int) * (k - 1));
        NE_ERROR_JMP(status, hipSuccess, NVSHMEMX_ERROR_INTERNAL, out, "hipMalloc failed\n");
        status = hipMemcpyHtoD((hipDeviceptr_t)((int **)step2_nbrs + i), &dev_ptr, sizeof(int *));
        NE_ERROR_JMP(status, hipSuccess, NVSHMEMX_ERROR_INTERNAL, out, "hipMemcpyHtoD failed\n");
    }

    status = hipMemcpyToSymbol(HIP_SYMBOL(reduce_recexch_step2_nbrs_d), &step2_nbrs, sizeof(void**));
    NE_ERROR_JMP(status, hipSuccess, NVSHMEMX_ERROR_INTERNAL, out, "memcopy to symbol failed \n");
    status = hipMemcpyToSymbol(HIP_SYMBOL(digit_d), &digit, sizeof(int *));
    NE_ERROR_JMP(status, hipSuccess, NVSHMEMX_ERROR_INTERNAL, out, "memcopy to symbol failed \n");

    nvshmemi_recexchalgo_get_neighbors(nvshmemi_state->mype, nvshmemi_state->npes);

    CUDA_CHECK(hipStreamSynchronize(0));

    goto fn_out;
out:
fn_out:
    return status;
}
