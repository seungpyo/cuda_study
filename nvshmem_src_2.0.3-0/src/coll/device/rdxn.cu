#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * See COPYRIGHT for license information
 */

#include "nvshmem.h"
#include "nvshmemx.h"
#include "gpu_coll.h"
#include "nvshmemi_team.h"
#include "nvshmemi_coll.h"

#ifdef __CUDA_ARCH__

#define GPU_HEAD_CHECKALL_OP(TYPENAME, TYPE, OP, dest, src, actual_src, nelems, start, stride, size) \
    do {                                                                                         \
        int i, j, k;                                                                             \
        int subelems = sizeof(TYPE) / sizeof(uint32_t);                                          \
        volatile uint32_t *header = NULL;                                                        \
        TYPE tmp;                                                                                \
        uint32_t *tmp_ptr = (uint32_t *)&tmp;                                                    \
        uint32_t *payload = NULL;                                                                \
        int my_active_set_pe = ((nvshmemi_mype_d - start) / stride);                             \
        TYPE *src_ptr = (TYPE *)actual_src;                                                      \
                                                                                                 \
        for (j = (my_active_set_pe - 1); j >= 0; j--) {                                          \
            for (i = 0; i < nelems; i++) {                                                       \
                for (k = 0; k < subelems; k++) {                                                 \
                    payload = (uint32_t *)((uint64_t *)src + (i * subelems) + k +                \
                                           (nelems * subelems * j));                             \
                    header = (uint32_t *)payload + 1;                                            \
                    while (1 != *header)                                                         \
                        ;                                                                        \
                    *header = 0;                                                                 \
                    *(tmp_ptr + k) = *payload;                                                   \
                }                                                                                \
                perform_gpu_rd_##OP(*((TYPE *)dest + i), *((TYPE *)src_ptr + i), tmp);           \
            }                                                                                    \
            src_ptr = dest;                                                                      \
        }                                                                                        \
        for (j = size - 1; j > my_active_set_pe; j--) {                                       \
            for (i = 0; i < nelems; i++) {                                                       \
                for (k = 0; k < subelems; k++) {                                                 \
                    payload = (uint32_t *)((uint64_t *)src + (i * subelems) + k +                \
                                           (nelems * subelems * j));                             \
                    header = (uint32_t *)payload + 1;                                            \
                    while (1 != *header)                                                         \
                        ;                                                                        \
                    *header = 0;                                                                 \
                    *(tmp_ptr + k) = *payload;                                                   \
                }                                                                                \
                perform_gpu_rd_##OP(*((TYPE *)dest + i), *((TYPE *)src_ptr + i), tmp);           \
            }                                                                                    \
            src_ptr = dest;                                                                      \
        }                                                                                        \
    } while (0)

#define GPU_LINEAR_REDUCE(TYPENAME, TYPE, OP, x, y, z, nelems)                         \
    do {                                                                               \
        int i;                                                                         \
        for (i = 0; i < nelems; i++) {                                                 \
            perform_gpu_rd_##OP(*((TYPE *)z + i), *((TYPE *)x + i), *((TYPE *)y + i)); \
        }                                                                              \
    } while (0)

#define GPU_RDXN_ON_DEMAND(TYPENAME, TYPE, OP, dest, source, nelems, start, stride, size, pWrk,     \
                           pSync)                                                                   \
    do {                                                                                            \
        int next_rank = -1;                                                                         \
        TYPE *op1 = NULL, *op2 = NULL;                                                              \
        int i;                                                                                      \
        volatile TYPE *tmp_operand;                                                                 \
        int my_active_set_pe = ((nvshmemi_mype_d - start) / stride);                                \
                                                                                                    \
        tmp_operand = (TYPE *) pWrk;                                                                \
                                                                                                    \
        nvshmem_##TYPENAME##_put((TYPE *)dest, (TYPE *)source, nelems, nvshmemi_mype_d);            \
        long counter = NVSHMEMI_SYNC_VALUE + 1;                                                     \
        for (i = 1; i < size; i++) {                                                                \
            next_rank = start + ((my_active_set_pe + i) % size) * stride;                           \
            nvshmem_##TYPENAME##_put_nbi((TYPE *)tmp_operand, (TYPE *)source, nelems, next_rank);   \
            nvshmemi_barrier(start, stride, size, pSync, &counter);                                 \
            op1 = (TYPE *)dest;                                                                     \
            op2 = (TYPE *)tmp_operand;                                                              \
            GPU_LINEAR_REDUCE(TYPENAME, TYPE, OP, op1, op2, op1, nelems);                           \
            nvshmemi_sync(start, stride, size, pSync, &counter);                                    \
        }                                                                                           \
        int end = start + size * stride;                                                            \
        for(i = 0; i < end; i++)                                                                    \
            pSync[i] = NVSHMEMI_SYNC_VALUE;                                                         \
    } while (0)

#define GPU_RDXN_SEGMENT(TYPENAME, TYPE, OP, dest, source, nelems, start, stride, size, pWrk, \
                         pSync)                                                                 \
    do {                                                                                        \
        int type_size = sizeof(TYPE);                                                           \
        int msg_len = nelems * type_size;                                                       \
        int next_rank = -1;                                                                     \
        TYPE *op1 = NULL, *op2 = NULL;                                                          \
        int i, j;                                                                               \
        volatile TYPE *tmp_operand;                                                             \
        int remainder = 0;                                                                      \
        int rnds_floor = 0;                                                                     \
        int offset = 0;                                                                         \
        int exchange_size = 0;                                                                  \
        int nvshm_gpu_rdxn_seg_size = NVSHMEMI_REDUCE_MIN_WRKDATA_SIZE;                        \
        int my_active_set_pe = ((nvshmemi_mype_d - start) / stride);                            \
                                                                                                \
        tmp_operand = (TYPE *) pWrk;                                                            \
        nvshmem_##TYPENAME##_put((TYPE *)dest, (const TYPE *)source, nelems, nvshmemi_mype_d);  \
                                                                                                \
        rnds_floor = msg_len / nvshm_gpu_rdxn_seg_size;                                         \
        remainder = msg_len % nvshm_gpu_rdxn_seg_size;                                          \
        long counter = 1;                                                                       \
        for (j = 0; j < rnds_floor; j++) {                                                      \
            exchange_size = nvshm_gpu_rdxn_seg_size;                                            \
            for (i = 1; i < size; i++) {                                                        \
                next_rank = start + ((my_active_set_pe + i) % size) * stride;                   \
                nvshmem_##TYPENAME##_put_nbi((TYPE *)tmp_operand, (TYPE *)source + offset,      \
                                         (exchange_size / sizeof(TYPE)), next_rank);            \
                nvshmemi_barrier(start, stride, size, pSync, &counter);                         \
                op1 = (TYPE *)dest + offset;                                                    \
                op2 = (TYPE *)tmp_operand;                                                      \
                GPU_LINEAR_REDUCE(TYPENAME, TYPE, OP, op1, op2, op1, (exchange_size / sizeof(TYPE)));     \
                nvshmemi_sync(start, stride, size, pSync, &counter);                            \
            }                                                                                   \
            offset += (exchange_size / sizeof(TYPE));                                           \
        }                                                                                       \
                                                                                                \
        if (remainder != 0) {                                                                   \
            exchange_size = remainder;                                                          \
            for (i = 1; i < size; i++) {                                                        \
                next_rank = start + ((my_active_set_pe + i) % size) * stride;                   \
                nvshmem_##TYPENAME##_put_nbi((TYPE *)tmp_operand, (const TYPE *)source + offset,          \
                                         (exchange_size / sizeof(TYPE)), next_rank);            \
                nvshmemi_barrier(start, stride, size, pSync, &counter);                         \
                op1 = (TYPE *)dest + offset;                                                    \
                op2 = (TYPE *)tmp_operand;                                                      \
                GPU_LINEAR_REDUCE(TYPENAME, TYPE, OP, op1, op2, op1, (exchange_size / sizeof(TYPE)));     \
                nvshmemi_sync(start, stride, size, pSync, &counter);                            \
            }                                                                                   \
        }                                                                                       \
        for(i = 0; i < NVSHMEMI_REDUCE_SYNC_SIZE; i++)                                          \
            pSync[i] = NVSHMEMI_SYNC_VALUE;                                                     \
    } while (0)

#define GPU_RDXN_ZCOPY_GET_BAR(TYPENAME, TYPE, OP, dest, source, nelems, start, stride, size,   \
                               pWrk, pSync)                                                       \
    do {                                                                                          \
        int next_rank = -1;                                                                       \
        int src_offset = -1;                                                                      \
        int next_offset = -1;                                                                     \
        char *base = NULL;                                                                        \
        char *peer_base = NULL;                                                                   \
        char *peer_source = NULL;                                                                 \
        int i;                                                                                    \
                                                                                                  \
        base = (char *)((void *)__ldg((const long long unsigned *)nvshmemi_peer_heap_base_d +      \
                                      nvshmemi_mype_d));                                           \
        src_offset = ((char *)source - base);                                                     \
                                                                                                  \
        next_rank = (nvshmemi_mype_d + (stride)) % (stride * size);                               \
        next_offset = src_offset;                                                                 \
        peer_base = (char *)((void *)__ldg((const long long unsigned *)nvshmemi_peer_heap_base_d + \
                                           next_rank));                                           \
        peer_source = peer_base + next_offset;                                                    \
        GPU_LINEAR_REDUCE(TYPENAME, TYPE, OP, (void *)source, peer_source, dest, nreduce);                  \
        long counter  = 1;                                                                        \
        nvshmemi_barrier(start, stride, size, pSync, &counter);                                   \
                                                                                                  \
        for (i = 2; i < size; i++) {                                                              \
            next_rank = (nvshmemi_mype_d + (i * stride)) % (stride * size);                       \
            next_offset = src_offset;                                                             \
            peer_base = (char *)((void *)__ldg(                                                   \
                (const long long unsigned *)nvshmemi_peer_heap_base_d + next_rank));               \
            peer_source = peer_base + next_offset;                                                \
            GPU_LINEAR_REDUCE(TYPENAME, TYPE, OP, dest, peer_source, dest, nreduce);                        \
            nvshmemi_barrier(start, stride, size, pSync, &counter);                               \
        }                                                                                         \
        for(i = 0; i < NVSHMEMI_REDUCE_SYNC_SIZE; i++)                                            \
            pSync[i] = NVSHMEMI_SYNC_VALUE;                                                       \
    } while (0)

#define NVSHMEMI_GPU_RDXN_PUTALL(TYPENAME, TYPE, OP, dest, source, nelems, start, stride, size,             \
                                 pWrk, pSync)                                                     \
    do {                                                                                          \
        int offset;                                                                               \
        int i, j;                                                                                 \
        int end = start + (stride * size);                                                        \
        uint32_t tmp[2];                                                                          \
        uint32_t payld;                                                                           \
        int subelems = sizeof(TYPE) / sizeof(uint32_t);                                           \
        int my_active_set_pe = ((nvshmemi_mype_d - start) / stride);                            \
        tmp[1] = 1;                                                                               \
                                                                                                  \
        for (j = 0; j < nelems * subelems; j++) {                                                 \
            payld = *((uint32_t *)source + j);                                                    \
            tmp[0] = payld;                                                                       \
            for (i = start; i < nvshmemi_mype_d; i += stride) {                                 \
                nvshmemx_long_signal((long *)pWrk + j + (nelems * subelems * my_active_set_pe),   \
                                     *((long *)tmp), i);                                          \
            }                                                                                     \
            for (i = nvshmemi_mype_d + stride; i < end; i += stride) {                          \
                nvshmemx_long_signal((long *)pWrk + j + (nelems * subelems * my_active_set_pe),   \
                                     *((long *)tmp), i);                                          \
            }                                                                                     \
        }                                                                                         \
        GPU_HEAD_CHECKALL_OP(TYPENAME, TYPE, OP, dest, pWrk, source, nelems, start, stride, size);    \
        __threadfence();                                                                          \
    } while (0)

#define NVSHMEMI_GPU_RDXN_PUTALL_DIRECT(TYPENAME, TYPE, OP, dest, source, nelems, start, stride,           \
                                        size, pWrk, pSync)                                       \
    do {                                                                                         \
        int offset;                                                                              \
        char *round_pwrk_dest;                                                                   \
        int i, j;                                                                                \
        int end = start + (stride * size);                                                       \
        uint32_t tmp[2];                                                                         \
        uint32_t payld;                                                                          \
        int subelems = sizeof(TYPE) / sizeof(uint32_t);                                          \
        int my_active_set_pe = ((nvshmemi_mype_d - start) / stride);                             \
        tmp[1] = 1;                                                                              \
        offset =                                                                                 \
            (char *)pWrk - (char *)(__ldg((const long long unsigned *)nvshmemi_peer_heap_base_d + \
                                          nvshmemi_mype_d));                                      \
                                                                                                 \
        for (j = 0; j < nelems * subelems; j++) {                                                \
            payld = *((uint32_t *)source + j);                                                   \
            tmp[0] = payld;                                                                      \
            for (i = nvshmemi_mype_d + stride; i < end; i += stride) {                           \
                round_pwrk_dest =                                                                \
                    (char *)(__ldg((const long long unsigned *)nvshmemi_peer_heap_base_d + i)) + \
                    offset;                                                                      \
                *((uint64_t *)round_pwrk_dest + j + (nelems * subelems * my_active_set_pe)) =    \
                    *((uint64_t *)tmp);                                                          \
            }                                                                                    \
            for (i = start; i < nvshmemi_mype_d; i += stride) {                                  \
                round_pwrk_dest =                                                                \
                    (char *)(__ldg((const long long unsigned *)nvshmemi_peer_heap_base_d + i)) +  \
                    offset;                                                                      \
                *((uint64_t *)round_pwrk_dest + j + (nelems * subelems * my_active_set_pe)) =    \
                    *((uint64_t *)tmp);                                                          \
            }                                                                                    \
        }                                                                                        \
        GPU_HEAD_CHECKALL_OP(TYPENAME, TYPE, OP, dest, pWrk, source, nelems, start, stride, size);         \
        __threadfence();                                                                         \
    } while (0)

#ifdef NVSHMEM_GPU_COLL_USE_LDST
#ifdef NVSHMEM_DISABLE_COLL_POLL
#define NVSHMEMI_GPU_RDXN(TYPENAME, TYPE, OP, dest, source, nelems, start, stride,csize, pWrk,   \
                          pSync)                                                                 \
    do {                                                                                         \
        GPU_RDXN_ZCOPY_GET_BAR(TYPENAME, TYPE, OP, dest, source, nelems, start, stride, size,    \
                               pWrk, pSync);                                                     \
    } while (0)
#else
#define NVSHMEMI_GPU_RDXN(TYPENAME, TYPE, OP, dest, source, nelems, start, stride, size, pWrk,   \
                          pSync)                                                                 \
    do {                                                                                         \
        int subelems = sizeof(TYPE) / sizeof(uint32_t);                                          \
        int pwrk_req_sz_allgather = ((subelems * nelems) * sizeof(uint64_t)) * size;             \
        /*int pwrk_req_sz_ring = ((subelems * nelems) * sizeof(uint64_t));*/                     \
        int wrk_size = NVSHMEMI_REDUCE_MIN_WRKDATA_SIZE * sizeof(TYPE);                          \
        if (subelems && pwrk_req_sz_allgather <= wrk_size) {                                     \
            NVSHMEMI_GPU_RDXN_PUTALL_DIRECT(TYPENAME, TYPE, OP, dest, source, nelems, start,               \
                                            stride, size, pWrk, pSync);                          \
        } else {                                                                                 \
            GPU_RDXN_ZCOPY_GET_BAR(TYPENAME, TYPE, OP, dest, source, nelems, start, stride,                \
                                   size, pWrk, pSync);                                           \
        }                                                                                        \
    } while (0)
#endif
#else
#define NVSHMEMI_GPU_RDXN(TYPENAME, TYPE, OP, dest, source, nelems, start, stride, size, pWrk,               \
                          pSync)                                                                   \
    do {                                                                                           \
        int k = gpu_coll_env_params_var_d.reduce_recexch_kval;                                     \
        if (start == 0 && stride == 1 && size == nvshmemi_npes_d &&                                \
            NVSHMEMI_REDUCE_MIN_WRKDATA_SIZE >=                                                    \
                ((k - 1) * nelems + k * reduce_recexch_step2_nphases_d * nelems + reduce_recexch_step2_nphases_d * nelems)) { \
            GPU_RDXN_RECEXCH(TYPENAME, TYPE, OP, dest, source, nelems, start, stride, size,                  \
                             pWrk, pSync);                                                         \
        } else {                                                                                   \
            if (NVSHMEMI_REDUCE_MIN_WRKDATA_SIZE >= (nelems * sizeof(TYPE))) {                     \
                GPU_RDXN_ON_DEMAND(TYPENAME, TYPE, OP, dest, source, nelems, start, stride,                  \
                                   size, pWrk, pSync);                                             \
            } else {                                                                               \
                GPU_RDXN_SEGMENT(TYPENAME, TYPE, OP, dest, source, nelems, start, stride, size,              \
                                 pWrk, pSync);                                                     \
            }                                                                                      \
        }                                                                                          \
    } while (0)
#endif


/* pWrk usage - (k - 1) * nreduce for step 1
              - k * step2_nphases * nreduce for receiving step 2 data
              - step2_nphases * nreduce for sending data of each phase */
#define GPU_RDXN_RECEXCH(TYPENAME, TYPE, OP, dst, source, nreduce, start, stride, size, pWrk,                \
                         pSync)                                                                    \
    do {                                                                                           \
        int step1_sendto = reduce_recexch_step1_sendto_d;                                                         \
        int step1_nrecvs = reduce_recexch_step1_nrecvs_d;                                                         \
        int *step1_recvfrom = reduce_recexch_step1_recvfrom_d;                                                    \
        int step2_nphases = reduce_recexch_step2_nphases_d;                                                       \
        int **step2_nbrs = reduce_recexch_step2_nbrs_d;                                                           \
        int rank = nvshmemi_mype_d;                                                                 \
        int k = gpu_coll_env_params_var_d.reduce_recexch_kval;                                     \
                                                                                                   \
        int in_step2 = (step1_sendto == -1); /* whether this rank participates in Step 2 */        \
                                                                                                   \
        if (in_step2 == 1) {                                                                       \
            for (int i = 0; i < nreduce; i++) {                                                    \
                dst[i] = source[i];                                                                \
            }                                                                                      \
        }                                                                                          \
                                                                                                   \
        if (in_step2 == 0) {                                                                       \
            int offset = (step1_sendto - rank - 1) * nreduce;                                      \
            nvshmem_##TYPENAME##_put_nbi(pWrk + offset, source, nreduce, step1_sendto);                \
            nvshmem_fence();                                                                       \
            nvshmemx_long_signal(pSync + rank, !NVSHMEMI_SYNC_VALUE, step1_sendto);                \
        } else if (step1_nrecvs != 0) {                                                            \
            nvshmem_long_wait_until_all(pSync + step1_recvfrom[step1_nrecvs - 1], step1_nrecvs,    \
                                        NULL, NVSHMEM_CMP_EQ, !NVSHMEMI_SYNC_VALUE);               \
            for (int i = 0; i < step1_nrecvs; i++) {                                               \
                int offset = (rank - step1_recvfrom[i] - 1) * nreduce;                             \
                GPU_LINEAR_REDUCE(TYPENAME, TYPE, OP, dst, (pWrk + offset), dst, nreduce);                   \
            }                                                                                      \
        }                                                                                          \
                                                                                                   \
        /* Step 2 */                                                                               \
        if (in_step2) {                                                                            \
            int send_offset = (k - 1) * nreduce + k * step2_nphases * nreduce;                     \
            int recv_offset = (k - 1) * nreduce;                                                   \
            for (int phase = 0; phase < step2_nphases; phase++) {                                  \
                int num_small = k - 1;                                                             \
                for (int i = 0; i < k - 1; i++) {                                                  \
                    if (step2_nbrs[phase][i] > rank) {                                             \
                        num_small = i;                                                             \
                        break;                                                                     \
                    }                                                                              \
                }                                                                                  \
                /* copy the data to end of pWrk that can be used as source for puts                \
                   while we use dst for reduction */                                               \
                for (int i = 0; i < nreduce; i++) {                                                \
                    pWrk[send_offset + phase * nreduce + i] = dst[i];                              \
                }                                                                                  \
                                                                                                   \
                for (int i = 0; i < k - 1; i++) {                                                  \
                    int offset = recv_offset + k * phase * nreduce + num_small * nreduce;          \
                    nvshmem_##TYPENAME##_put_nbi(pWrk + offset, pWrk + send_offset + phase * nreduce,  \
                                             nreduce, step2_nbrs[phase][i]);                       \
                }                                                                                  \
                nvshmem_fence();                                                                   \
                for (int i = 0; i < k - 1; i++) {                                                  \
                    nvshmemx_long_signal(pSync + rank, NVSHMEMI_SYNC_VALUE + 1,                    \
                                         step2_nbrs[phase][i]);                                    \
                }                                                                                  \
                                                                                                   \
                for (int i = 0; i < k - 1; i++) {                                                  \
                    nvshmem_long_wait_until(pSync + step2_nbrs[phase][i], NVSHMEM_CMP_EQ,          \
                                            NVSHMEMI_SYNC_VALUE + 1);                              \
                    int offset = recv_offset + k * phase * nreduce;                                \
                    if (step2_nbrs[phase][i] < rank)                                               \
                        offset += i * nreduce;                                                     \
                    else                                                                           \
                        offset += (i + 1) * nreduce;                                               \
                    GPU_LINEAR_REDUCE(TYPENAME, TYPE, OP, dst, (pWrk + offset), dst, nreduce);               \
                }                                                                                  \
                /*nvshmem_quiet(); */ /*wait for my puts to complete */                            \
            }                                                                                      \
        }                                                                                          \
                                                                                                   \
        /* Step 3 */                                                                               \
        if (step1_nrecvs > 0) {                                                                    \
            for (int i = 0; i < step1_nrecvs; i++) {                                               \
                nvshmem_##TYPENAME##_put_nbi(dst, dst, nreduce, step1_recvfrom[i]);                    \
                nvshmem_fence();                                                                   \
                nvshmemx_long_signal(pSync + rank, NVSHMEMI_SYNC_VALUE + 1, step1_recvfrom[i]);    \
            }                                                                                      \
        } else if (step1_sendto != -1) {                                                           \
            nvshmem_long_wait_until(pSync + step1_sendto, NVSHMEM_CMP_EQ, NVSHMEMI_SYNC_VALUE + 1);\
        }                                                                                          \
                                                                                                   \
        for (int i = 0; i < nvshmemi_npes_d; i++)                                                   \
            pSync[i] = NVSHMEMI_SYNC_VALUE; /* should this be a volatile write? */                  \
                                                                                                   \
    } while (0);

#define DEFN_NVSHMEMI_TYPENAME_OP_REDUCE(TYPENAME, TYPE, OP)                                             \
    __device__ void nvshmemi_##TYPENAME##_##OP##_reduce(TYPE *dest, const TYPE *source, int nreduce,     \
                                    int start, int stride, int size, TYPE * pWrk, long *pSync) {         \
        NVSHMEMI_GPU_RDXN(TYPENAME, TYPE, OP, dest, source, nreduce, start, stride, size, pWrk, pSync);  \
    }

NVSHMEMI_REPT_FOR_BITWISE_REDUCE_TYPES(DEFN_NVSHMEMI_TYPENAME_OP_REDUCE, and)
NVSHMEMI_REPT_FOR_BITWISE_REDUCE_TYPES(DEFN_NVSHMEMI_TYPENAME_OP_REDUCE, or)
NVSHMEMI_REPT_FOR_BITWISE_REDUCE_TYPES(DEFN_NVSHMEMI_TYPENAME_OP_REDUCE, xor)

NVSHMEMI_REPT_FOR_STANDARD_REDUCE_TYPES(DEFN_NVSHMEMI_TYPENAME_OP_REDUCE, max)
NVSHMEMI_REPT_FOR_STANDARD_REDUCE_TYPES(DEFN_NVSHMEMI_TYPENAME_OP_REDUCE, min)

NVSHMEMI_REPT_FOR_ARITH_REDUCE_TYPES(DEFN_NVSHMEMI_TYPENAME_OP_REDUCE, sum)
NVSHMEMI_REPT_FOR_ARITH_REDUCE_TYPES(DEFN_NVSHMEMI_TYPENAME_OP_REDUCE, prod)

#undef DEFN_NVSHMEMI_TYPENAME_OP_REDUCE

#define DEFN_NVSHMEM_TYPENAME_OP_REDUCE(TYPENAME, TYPE, OP)                                                         \
    __device__ int nvshmem_##TYPENAME##_##OP##_reduce(nvshmem_team_t team, TYPE *dest, const TYPE *source, size_t nreduce) { \
        nvshmemi_team_t *teami = nvshmemi_team_pool_d[team];                                                        \
        TYPE *pWrk = (TYPE *)nvshmemi_team_get_psync(teami, REDUCE);                                                \
        long *pSync = (long *)((long *)pWrk + NVSHMEMI_REDUCE_MIN_WRKDATA_SIZE);                                    \
        nvshmem_barrier(team);                                                                                      \
        nvshmemi_##TYPENAME##_##OP##_reduce(dest, source, nreduce, teami->start, teami->stride, teami->size, pWrk,  \
                          pSync);                                                                                   \
        return 0;                                                                                                   \
    }

NVSHMEMI_REPT_FOR_BITWISE_REDUCE_TYPES(DEFN_NVSHMEM_TYPENAME_OP_REDUCE, and)
NVSHMEMI_REPT_FOR_BITWISE_REDUCE_TYPES(DEFN_NVSHMEM_TYPENAME_OP_REDUCE, or)
NVSHMEMI_REPT_FOR_BITWISE_REDUCE_TYPES(DEFN_NVSHMEM_TYPENAME_OP_REDUCE, xor)

NVSHMEMI_REPT_FOR_STANDARD_REDUCE_TYPES(DEFN_NVSHMEM_TYPENAME_OP_REDUCE, max)
NVSHMEMI_REPT_FOR_STANDARD_REDUCE_TYPES(DEFN_NVSHMEM_TYPENAME_OP_REDUCE, min)

NVSHMEMI_REPT_FOR_ARITH_REDUCE_TYPES(DEFN_NVSHMEM_TYPENAME_OP_REDUCE, sum)
NVSHMEMI_REPT_FOR_ARITH_REDUCE_TYPES(DEFN_NVSHMEM_TYPENAME_OP_REDUCE, prod)

#undef DEFN_NVSHMEM_TYPENAME_OP_REDUCE
#endif
