#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * See COPYRIGHT for license information
 */

#include "nvshmem.h"
#include "nvshmemx.h"
#include "gpu_coll.h"
#include "nvshmemi_coll.h"
#include <cstdio>
#include <cassert>

#ifdef __CUDA_ARCH__

#define NVSHMEMI_GPU_COLLECT_THREADGROUP_ALLPUSH(SC, SC_SUFFIX, SC_PREFIX, TYPENAME, TYPE, dest, source, \
                                                 nelems, PE_start, PE_stride, PE_size, pSync)     \
    do {                                                                                          \
        int stride = PE_stride;                                                                   \
        int next_rank;                                                                            \
        int next_offset;                                                                          \
        const int mype = nvshmemi_mype_d;                                                         \
        int my_idx_in_active_set = (mype - PE_start) / PE_stride;                                 \
                                                                                                  \
        for (int ii = 0; ii < PE_size; ii++) {                                                    \
            next_rank = PE_start + ((my_idx_in_active_set + ii) % PE_size) * stride;              \
            next_offset = nelems * ((mype - PE_start) / stride);                                  \
            nvshmem##SC_PREFIX##_##TYPENAME##_put_nbi##SC_SUFFIX(dest + next_offset,              \
                                                                 source, nelems, next_rank);      \
        }                                                                                         \
        nvshmem##SC_PREFIX##i_barrier##SC_SUFFIX(PE_start, PE_stride, PE_size, pSync, NULL);      \
    } while (0)

#define NVSHMEMI_GPU_COLLECT_THREADGROUP_P2P_ALLPUSH(SC, SC_SUFFIX, SC_PREFIX, TYPENAME, TYPE, dest, source, \
                                                    nelems, PE_start, PE_stride, PE_size, pSync) \
    do {                                                                                         \
        int stride = PE_stride;                                                                  \
        int next_rank;                                                                           \
        int next_offset;                                                                         \
        const int mype = nvshmemi_mype_d;                                                        \
        int my_idx_in_active_set = (mype - PE_start) / PE_stride;                                \
        TYPE *dst_ptr;                                                                           \
        NVSHMEMI_DECL_THREAD_IDX_##SC();                                                         \
        NVSHMEMI_DECL_THREADGROUP_SIZE_##SC();                                                   \
        for (int ii = 0; ii < PE_size; ii++) {                                                   \
            next_rank = PE_start + ((my_idx_in_active_set + ii) % PE_size) * stride;             \
            next_offset = nelems * ((mype - PE_start) / stride);                                 \
            dst_ptr = (TYPE *)nvshmem_ptr((void *)(dest + next_offset), next_rank);              \
            GPU_BITS_COPY_THREADGROUP_DIRECT(TYPENAME, TYPE, dst_ptr, source,                    \
                                             nelems, myIdx, groupSize);                          \
        }                                                                                        \
        nvshmem##SC_PREFIX##i_barrier##SC_SUFFIX(PE_start, PE_stride, PE_size, pSync, NULL);     \
    } while (0)


#define NVSHMEMI_GPU_COLLECT_THREADGROUP(SC, SC_SUFFIX, SC_PREFIX, TYPENAME, TYPE, dest, source, \
                                         nelems, PE_start, PE_stride, PE_size, pSync)            \
    do {                                                                                         \
        if (nvshmemi_job_connectivity_d <= NVSHMEMI_JOB_GPU_LDST)                                \
            NVSHMEMI_GPU_COLLECT_THREADGROUP_P2P_ALLPUSH(SC, SC_SUFFIX, SC_PREFIX, TYPENAME, TYPE,  \
                                                         dest, source, nelems, PE_start, PE_stride, \
                                                         PE_size, pSync);                        \
        else                                                                                     \
            NVSHMEMI_GPU_COLLECT_THREADGROUP_ALLPUSH(SC, SC_SUFFIX, SC_PREFIX, TYPENAME, TYPE,   \
                                                     dest, source, nelems, PE_start, PE_stride,  \
                                                     PE_size, pSync);                            \
    } while (0)


#define DEFN_NVSHMEMXI_TYPENAME_COLLECT_THREADGROUP(SC, SC_SUFFIX, SC_PREFIX, TYPENAME, TYPE)                      \
    __device__ void nvshmem##SC_PREFIX##i_##TYPENAME##_collect##SC_SUFFIX(TYPE *dest, const TYPE *source, size_t nelems, \
                                                        int PE_start, int PE_stride, int PE_size, long *pSync) {   \
        NVSHMEMI_GPU_COLLECT_THREADGROUP(SC, SC_SUFFIX, SC_PREFIX, TYPENAME, TYPE, dest, source,                   \
                                         nelems, PE_start, PE_stride, PE_size, pSync);                             \
    }

NVSHMEMI_REPT_FOR_STANDARD_RMA_TYPES_WITH_SCOPE2(DEFN_NVSHMEMXI_TYPENAME_COLLECT_THREADGROUP, thread, , )
NVSHMEMI_REPT_FOR_STANDARD_RMA_TYPES_WITH_SCOPE2(DEFN_NVSHMEMXI_TYPENAME_COLLECT_THREADGROUP, warp, _warp, x)
NVSHMEMI_REPT_FOR_STANDARD_RMA_TYPES_WITH_SCOPE2(DEFN_NVSHMEMXI_TYPENAME_COLLECT_THREADGROUP, block, _block, x)
#undef DEFN_NVSHMEMXI_TYPENAME_COLLECT_THREADGROUP


#define DEFN_NVSHMEMX_TYPENAME_COLLECT_THREADGROUP(SC, SC_SUFFIX, SC_PREFIX, TYPENAME, TYPE)               \
    __device__ int nvshmem##SC_PREFIX##_##TYPENAME##_collect##SC_SUFFIX(nvshmem_team_t team, TYPE *dest,   \
                                                                  const TYPE *source, size_t nelems) {     \
        nvshmemi_team_t *teami = nvshmemi_team_pool_d[team];                                               \
        nvshmem##SC_PREFIX##_barrier##SC_SUFFIX(team);                                                     \
        nvshmem##SC_PREFIX##i_##TYPENAME##_collect##SC_SUFFIX(dest, source, nelems, teami->start, teami->stride,\
                                            teami->size, nvshmemi_team_get_psync(teami, COLLECT));         \
        return 0;                                                                                          \
    }

NVSHMEMI_REPT_FOR_STANDARD_RMA_TYPES_WITH_SCOPE2(DEFN_NVSHMEMX_TYPENAME_COLLECT_THREADGROUP, thread, , )
NVSHMEMI_REPT_FOR_STANDARD_RMA_TYPES_WITH_SCOPE2(DEFN_NVSHMEMX_TYPENAME_COLLECT_THREADGROUP, warp, _warp, x)
NVSHMEMI_REPT_FOR_STANDARD_RMA_TYPES_WITH_SCOPE2(DEFN_NVSHMEMX_TYPENAME_COLLECT_THREADGROUP, block, _block, x)
#undef DEFN_NVSHMEMX_TYPENAME_COLLECT_THREADGROUP

#endif

#define COLLECT_ON_STREAM_KERNEL(TYPENAME, TYPE)                                                   \
    __global__ void collect_##TYPENAME##_on_stream_kernel(TYPE *dest, const TYPE *source,          \
                                                          size_t nelems, int PE_start,             \
                                                          int PE_stride, int PE_size, long *pSync) {    \
        if (!blockIdx.x)                                                                           \
            nvshmemxi_##TYPENAME##_collect_block(dest, source, nelems, PE_start, PE_stride, PE_size,\
                                           pSync);                                                 \
    }

NVSHMEMI_REPT_FOR_STANDARD_RMA_TYPES(COLLECT_ON_STREAM_KERNEL)
#undef COLLECT_ON_STREAM_KERNEL

#define CALL_COLLECT_ON_STREAM(TYPENAME, TYPE)                                                     \
    extern "C" void call_##TYPENAME##_collect_on_stream_kern(                                      \
        TYPE *dest, const TYPE *source, size_t nelems, int PE_start, int PE_stride,                \
        int PE_size, long *pSync, hipStream_t stream) {                                           \
        int num_threads_per_block = (MAX_THREADS_PER_CTA > nelems) ? nelems : MAX_THREADS_PER_CTA; \
        int num_blocks = 1;                                                                        \
        collect_##TYPENAME##_on_stream_kernel<<<num_blocks, num_threads_per_block, 0, stream>>>(   \
            dest, source, nelems, PE_start, PE_stride, PE_size, pSync);                            \
        CUDA_RUNTIME_CHECK(hipGetLastError());                                                    \
    }

NVSHMEMI_REPT_FOR_STANDARD_RMA_TYPES(CALL_COLLECT_ON_STREAM)
#undef CALL_COLLECT_ON_STREAM
