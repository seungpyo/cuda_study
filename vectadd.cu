#include "hip/hip_runtime.h"
#include <iostream>
#include <cunistd>
#include "cuutils.h"

#define N 8
#define CUUTIL_DEBUG
using namespace std;

__global__ void vect_add(int *c, int *a, int *b) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	for(; tid < N; tid += blockDim.x * gridDim.x) {

		c[tid] = a[tid] + b[tid];
	}
}


int main() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	for(int i = 0; i < N; ++i) {
		a[i] = -1;
		b[i] = -2;
		c[i] = -10;
	}
	CUUTIL_ERRCHK( hipMalloc( (void**)&dev_a, N*sizeof(int) ) );
	CUUTIL_ERRCHK( hipMalloc( (void**)&dev_b, N*sizeof(int) ) );
	CUUTIL_ERRCHK( hipMalloc( (void**)&dev_c, N*sizeof(int) ) );
	CUUTIL_ERRCHK( hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice) );
	CUUTIL_ERRCHK( hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice) );
	CUUTIL_ERRCHK( hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice) );
	
	vect_add<<<1, N>>>(dev_c, dev_a, dev_b);
	CUUTIL_ERRCHK( hipMemcpy(a, dev_a, N*sizeof(int), hipMemcpyDeviceToHost) );
	CUUTIL_ERRCHK( hipMemcpy(b, dev_b, N*sizeof(int), hipMemcpyDeviceToHost) );
	CUUTIL_ERRCHK( hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost) );
	cout << "a: "; for(int i = 0; i < N; ++i) cout << a[i] << ", "; cout << endl;
	cout << "b: "; for(int i = 0; i < N; ++i) cout << b[i] << ", "; cout << endl;
	cout << "c: "; for(int i = 0; i < N; ++i) cout << c[i] << ", "; cout << endl;

	

	return 0;
}
